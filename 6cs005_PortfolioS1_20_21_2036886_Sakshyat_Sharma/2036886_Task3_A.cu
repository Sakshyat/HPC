#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/******************************************************************************

  Sakshyat Sharma, 2036886
  
  Task3_A:
  CUDA Password Encrypt and Decrypt

  Compile with:
    nvcc -o 2036886_Task3_A 2036886_Task3_A.cu
  
  Run with:
    ./2036886_Task3_A

******************************************************************************/

//__global__ --> GPU function which can be launched by many blocks and threads
//__device__ --> GPU function or variables
//__host__ --> CPU function or variables

const char * passInput = "tpsdwy2712";  //encrypted password

__device__ char* CudaCrypt(char* rawPassword){

	char * newPassword = (char *) malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all lower case letter limits
			if(newPassword[i] > 122){
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ //checking number section
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword;
}

__global__ void crack(char * alphabet, char * numbers, char * passInput){

char genRawPass[4];

genRawPass[0] = alphabet[blockIdx.x];
genRawPass[1] = alphabet[blockIdx.y];

genRawPass[2] = numbers[threadIdx.x];
genRawPass[3] = numbers[threadIdx.y];

//firstLetter - 'a' - 'z' (26 characters)
//secondLetter - 'a' - 'z' (26 characters)
//firstNum - '0' - '9' (10 characters)
//secondNum - '0' - '9' (10 characters)

//Idx --> gives current index of the block or thread

//check


char * enc_pw = CudaCrypt(genRawPass);
char * pass_input = passInput;
int  counter = 0;

for(int i=0;i<=9;i++){

	if(i >= 0 && i < 6){ 
		if(enc_pw[i] == pass_input[i]){
		counter = counter+1;
		}
	}
	else if(counter == 9){
	  printf("Encrypted password %s = %c %c %c %c\n",CudaCrypt(genRawPass), genRawPass[0],genRawPass[1],genRawPass[2],genRawPass[3]);
	}
	else{ 
		if((int)enc_pw[i] == (int)pass_input[i]){
		counter = counter+1;
		}
	}
     }
}

int time_difference(struct timespec *start, struct timespec *finish, long long int *difference){
long long int ds = finish->tv_sec - start->tv_sec;
long long int dn = finish->tv_nsec - start->tv_nsec;

if(dn < 0 ){
ds--;
dn += 1000000000;
}
*difference = ds * 1000000000 + dn;
return !(*difference > 0);
}

int main(int argc, char ** argv){

char cpuAlphabet[26] = {'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z'};

char cpuNumbers[26] = {'0','1','2','3','4','5','6','7','8','9'};

char * gpuPass;
hipMalloc( (void**) &gpuPass, sizeof(char *) * 11); 
hipMemcpy(gpuPass, passInput, sizeof(char *) * 11, hipMemcpyHostToDevice);

char * gpuAlphabet;
hipMalloc( (void**) &gpuAlphabet, sizeof(char) * 26); 
hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);

char * gpuNumbers;
hipMalloc( (void**) &gpuNumbers, sizeof(char) * 26); 
hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 26, hipMemcpyHostToDevice);


struct timespec start, finish;
long long int time_elapsed;

clock_gettime(CLOCK_MONOTONIC, &start);

crack<<< dim3(26,26,1), dim3(10,10,1) >>>( gpuAlphabet, gpuNumbers,gpuPass);
hipDeviceSynchronize();

clock_gettime(CLOCK_MONOTONIC, &finish);
time_difference(&start, &finish, &time_elapsed);
printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
           (time_elapsed / 1.0e9));

return 0;
}













